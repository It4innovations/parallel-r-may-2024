#include "hip/hip_runtime.h"
/** original code from https://developer.nvidia.com/blog/introduction-cuda-dynamic-parallelism/  */
/** @file histo-global.cu histogram with global memory atomics */
#include </apps/all/R/4.1.0-foss-2021a/lib/R/library/Rcpp/include/Rcpp.h>
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <time.h>
using namespace Rcpp;

/** CUDA check macro */
#define cucheck(call)                                                         \
{                                                                             \
  hipError_t res = (call);                                                   \
  if(res != hipSuccess) {                                                    \
    const char* err_str = hipGetErrorString(res);                            \
    fprintf(stderr, "%s (%d): %s in %s", __FILE__, __LINE__, err_str, #call);	\
    exit(-1);                                                                 \
  }                                                                           \
}

/** a useful function to compute the number of threads */
int divup(int x, int y) { return x / y + (x % y ? 1 : 0); }

/** a simple complex type */
struct complex {
  __host__ __device__ complex(float re, float im = 0) {
    this->re = re;
    this->im = im;
  }
  /** real and imaginary part */
  float re, im;
}; // struct complex

// operator overloads for complex numbers
inline __host__ __device__ complex operator+
(const complex &a, const complex &b) {
  return complex(a.re + b.re, a.im + b.im);
}
inline __host__ __device__ complex operator-
(const complex &a) { return complex(-a.re, -a.im); }
inline __host__ __device__ complex operator-
(const complex &a, const complex &b) {
  return complex(a.re - b.re, a.im - b.im);
}
inline __host__ __device__ complex operator*
(const complex &a, const complex &b) {
  return complex(a.re * b.re - a.im * b.im, a.im * b.re + a.re * b.im);
}
inline __host__ __device__ float abs2(const complex &a) {
  return a.re * a.re + a.im * a.im;
}
inline __host__ __device__ complex operator/
(const complex &a, const complex &b) {
  float invabs2 = 1 / abs2(b);
  return complex((a.re * b.re + a.im * b.im) * invabs2,
                 (a.im * b.re - b.im * a.re) * invabs2);
}  // operator/

#define MAX_DWELL 1024
#define BS 1024
/** computes the dwell for a single pixel */
__device__ int pixel_dwell
(int w, int h, complex cmin, complex cmax, int x, int y, int max_dwell) {
  complex dc = cmax - cmin;
  float fx = (float)x / w, fy = (float)y / h;
  complex c = cmin + complex(fx * dc.re, fy * dc.im);
  int dwell = 0;
  complex z = c;
  while(dwell < max_dwell && abs2(z) < 2 * 2) {
    z = z * z + c;
    dwell++;
  }
  return dwell;
}  // pixel_dwell

/** computes the dwells for Mandelbrot image 
 @param dwells the output array
 @param w the width of the output image
 @param h the height of the output image
 @param cmin the complex value associated with the left-bottom corner of the
 image
 @param cmax the complex value associated with the right-top corner of the
 image
 */
__global__ void mandelbrot_k
(int *dwells, int w, int h, complex cmin, complex cmax, int max_dwell) {
  // complex value to start iteration (c)
  int x = threadIdx.x + blockIdx.x * blockDim.x;
  int y = threadIdx.y + blockIdx.y * blockDim.y;
  int dwell = pixel_dwell(w, h, cmin, cmax, x, y, max_dwell);
  dwells[y * w + x] = dwell;
}  // mandelbrot_k


std::vector<int> mandelbrot_main
(int wh, int max_dwell) {
  // allocate memory
  int w = wh, h = wh;
  size_t dwell_sz = w * h * sizeof(int);
  int *h_dwells, *d_dwells;
  cucheck(hipMalloc((void**)&d_dwells, dwell_sz));
  h_dwells = (int*)malloc(dwell_sz);
  
  // compute the dwells, copy them back
  dim3 bs(64, 4), grid(divup(w, bs.x), divup(h, bs.y));
  // Call the CUDA implementation of the mandelbrot function from Kernels.h with signature:
  mandelbrot_k<<<grid, bs>>>
    (d_dwells, w, h, complex(-1.5, -1), complex(0.5, 1), max_dwell);
  cucheck(hipDeviceSynchronize());
  cucheck(hipMemcpy(h_dwells, d_dwells, dwell_sz, hipMemcpyDeviceToHost));
  
  // Convert the resultant array to a vector
  std::vector<int> res(h_dwells, h_dwells + (w * h));
  
  // free data
  hipFree(d_dwells);
  free(h_dwells);
  return res;
}
